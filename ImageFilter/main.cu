#include "main.cuh"

int main(int argc, char** argv)
{
    CUDAf::getDeviceName();
    printf("Use cuda device 0\n");

    unsigned char* buffer = nullptr;

    printf("Reading Image\n");
    cv::Mat image = cvf::cvReadImg("Enhanced_CPP_output.png", buffer);

    printf("Image info :\nWidth = %d\nHeight = %d\nChannel = %d\n", image.cols, image.rows, image.channels());

    hipError_t error;

    unsigned char* dInput = nullptr;
    unsigned char* dOutput = nullptr;
    size_t buffer_size = image.rows * image.cols * image.channels();
    
    // Transfer Mem to GPU
    error = hipMalloc(&dInput, buffer_size);
    if (error != hipSuccess) {
        printf("Error dInput hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMemcpy(dInput, buffer, buffer_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dInput hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dOutput, buffer_size);
    if (error != hipSuccess) {
        printf("Error dOutput hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Invoke Kernel
    dim3 dimBlock = {16, 8, 3};
    dim3 dimGrid;
    dimGrid.x = (image.cols + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (image.rows + dimBlock.y - 1) / dimBlock.y;
    dimGrid.z = (image.channels() + dimBlock.z - 1) / dimBlock.z;

    //printf("dimGrid [%d, %d, %d]\n", dimGrid.x, dimGrid.y, dimGrid.z);
    
    CUDAf::dMeanFilter<<<dimGrid, dimBlock>>>(image.rows, image.cols, image.rows, image.cols, 5, 2, image.channels(), dInput, dOutput);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error Kernel : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Transfer Back to CPU
    error = hipMemcpy(buffer, dOutput, buffer_size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error buffer hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    printf("Output Image\n");
    cvf::cvOutputImg("dMeanFilterOutput.png", image, buffer);

    if (buffer != nullptr) {
        delete[] buffer;
    }

    if (dInput != nullptr) {
        error = hipFree(dInput);
        if (error != hipSuccess) {
            printf("Error dInput hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
    }

    if (dOutput != nullptr) {
        error = hipFree(dOutput);
        if (error != hipSuccess) {
            printf("Error dOutput hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
    }

    return 0;
}
